#include "hip/hip_runtime.h"
// This file contains kernels to compute the steinhardt parameters and its gradient


/**
 * Perform the first step of computing q6.
 */

 __device__ real reduceValue(real value, volatile real* temp) {
     const int thread = threadIdx.x;
     __syncthreads();
     temp[thread] = value;
     __syncthreads();
     for (unsigned int step = 1; step < 32; step *= 2) {
         if (thread+step < blockDim.x && thread%(2*step) == 0)
             temp[thread] = temp[thread] + temp[thread+step];
         SYNC_WARPS
     }
     for (unsigned int step = 32; step < blockDim.x; step *= 2) {
         if (thread+step < blockDim.x && thread%(2*step) == 0)
             temp[thread] = temp[thread] + temp[thread+step];
         __syncthreads();
     }
     return temp[0];
 }


__device__ real legendre(real rdot,int steinhardt_order){
	 real result=0;

	 if(steinhardt_order == 6){
	        real pow2=rdot*rdot;
		real pow4=pow2*pow2;
		real pow6=pow4*pow2;
	 	result= (231*pow6-315*pow4+105*pow2-5)/16;
	
		}
	 else if (steinhardt_order == 4){
	      	real pow2=rdot*rdot;
		real pow4=pow2*pow2;
	 	result=(35*pow4-30*pow2+3)/8;
	}
	 return result;
}

__device__ real legendre_deriv(real rdot, int steinhardt_order){
	 real result=0;
	 if(steinhardt_order==6){
		real pow3=rdot*rdot*rdot;
		real pow5=pow3*rdot*rdot;
		
	 	result=(1386*pow5-1260*pow3+210*rdot)/16;
		}
	 else if(steinhardt_order == 4){
	      	real pow3=rdot*rdot*rdot;
	 	result=(140*pow3-60*rdot)/8;
		}
	 return result;
}

extern "C" __global__ void computeSteinhardt(int numParticles, const real4* __restrict__ posq,
         const int* __restrict__ particles, real* buffer, unsigned long long* __restrict__ forceBuffers, int paddedNumAtoms, real4 periodicBoxSize, real4 invPeriodicBoxSize,
                 real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ,real* __restrict__ M, real* __restrict__ N, real* __restrict__ F) {
    extern __shared__ volatile real temp[];


		unsigned int index = blockIdx.x*blockDim.x+threadIdx.x;
		while(index < numParticles){
			real3 positioni=trimTo3(posq[particles[index]]);
			real sumN=0;
			real sumM=0;
			F[3*index]=0;
			F[3*index+1]=0;
			F[3*index+2]=0;
			for(int j=0; j<numParticles; j++){
				if( j!=index ){
					real3 positionj=trimTo3(posq[particles[j]]);
					real3 rij= make_real3(positioni.x-positionj.x, positioni.y-positionj.y, positioni.z-positionj.z);
					APPLY_PERIODIC_TO_DELTA(rij);
					real rij_norm=sqrtf(rij.x*rij.x + rij.y*rij.y + rij.z*rij.z);
					if(rij_norm<1.6*CUTOFF){
						//real rij_pow6=powf((rij_norm-CUTOFF)/1,6);
						//real switch_ij=(1-rij_pow6)/(1-powf(rij_pow6,2));
						real switch_ij=(1-tanhf((rij_norm-CUTOFF)/0.2));
						sumN += switch_ij;
						for(int k=0; k<numParticles; k++){
							if (k != index){
								real3 positionk=trimTo3(posq[particles[k]]);
								real3 rik= make_real3(positioni.x-positionk.x, positioni.y-positionk.y, positioni.z-positionk.z);
								APPLY_PERIODIC_TO_DELTA(rik);
								real rik_norm=sqrtf(rik.x*rik.x + rik.y*rik.y + rik.z*rik.z);
								if(rik_norm<1.6*CUTOFF){
									//real rik_pow6=powf((rik_norm-CUTOFF)/1,6);
									//real switch_ik=(1-rik_pow6)/(1-powf(rik_pow6,2));

									real switch_ik=(1-tanhf((rik_norm-CUTOFF)/0.2));
									real rdot = (rij.x*rik.x + rij.y*rik.y + rij.z*rik.z)/(rik_norm*rij_norm);
									//real P6=(231*powf(rdot,6.0)-315*powf(rdot,4.0)+105*powf(rdot,2.0)-5)/16;
									real P6=legendre(rdot, STEINHARDT_ORDER);
									//M[i] += P6*switch_ik*switch_ij;
									sumM += P6*switch_ik*switch_ij;
									//printf("%d %d %d M %f p %f sik %f sij %f rd %f\n", index, j, k, sumM, P6, switch_ik, switch_ij, rdot);
								}
							}
						}
					}
				}
			}
			M[index]=sumM;
			N[index]=sumN;
			//printf("%f %f\n", sumM,sumN);
			index += blockDim.x*gridDim.x;
		}


}

/**
 * Compute forces by calculating the derivative
*/
extern "C" __global__ void computeSteinhardtForces(int numParticles, const real4* __restrict__ posq,
	const int* __restrict__ particles, real* buffer, unsigned long long* __restrict__ forceBuffers, int paddedNumAtoms, real4 periodicBoxSize, real4 invPeriodicBoxSize,
	real4 periodicBoxVecX, real4 periodicBoxVecY, real4 periodicBoxVecZ,real* __restrict__ M, real* __restrict__ N, real* F, real Q_tot) {
		extern __shared__ volatile real temp[];
		unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
		real prefactor=sqrtf(4*3.14159/(2*STEINHARDT_ORDER+1))/numParticles;


		while(i < numParticles){

			real3 positioni=trimTo3(posq[particles[i]]);

			real3 Ficomp=make_real3(0);
      real3 Ficomp2=make_real3(0);
			real M_prefactor=-sqrtf(4*3.14159/((2*STEINHARDT_ORDER+1)*M[i]))/(2*numParticles*N[i]);
			real N_prefactor=-sqrtf(M[i]*4*3.14159/(2*STEINHARDT_ORDER+1))/(N[i]*N[i]*numParticles);
			//printf("%f %f %f %f\n",M_prefactor,N_prefactor, M[i], N[i]);

			for(int j=0; j<numParticles; j++){
        real3 Fjcomp2=make_real3(0);
				if( j!=i ){
					real3 positionj=trimTo3(posq[particles[j]]);
					real3 rij= make_real3(positioni.x-positionj.x, positioni.y-positionj.y, positioni.z-positionj.z);
					APPLY_PERIODIC_TO_DELTA(rij);
					real rij_norm=sqrtf(rij.x*rij.x + rij.y*rij.y + rij.z*rij.z);
					if(rij_norm < 2.0*CUTOFF){

						real3 delta_rij_norm=rij/(2*rij_norm);

						real sech_ij=1/coshf((rij_norm-CUTOFF)/0.2);
						real switch_ij = (1-tanhf((rij_norm-CUTOFF)/0.2));
						real delta_switch_ij = -5*sech_ij*sech_ij;

						real3 Fjcomp=make_real3(0);
						Fjcomp2 = -delta_switch_ij*delta_rij_norm;
						Ficomp2 += delta_switch_ij*delta_rij_norm;

						for(int k=0; k<numParticles; k++){
							if(k!=i){
								real3 positionk=trimTo3(posq[particles[k]]);
								real3 rik= make_real3(positioni.x-positionk.x, positioni.y-positionk.y, positioni.z-positionk.z);
								APPLY_PERIODIC_TO_DELTA(rik);

								real rik_norm=sqrtf(rik.x*rik.x + rik.y*rik.y + rik.z*rik.z);
								if(rik_norm < 2.0*CUTOFF){
									real3 delta_rik_norm=rik/(2*rik_norm);

									real switch_ik = (1-tanhf((rik_norm-CUTOFF)/0.2));
									real sech_ik=1/coshf((rik_norm-CUTOFF)/0.2);
									real delta_switch_ik = -5*sech_ik*sech_ik;

									real rdot = (rij.x*rik.x + rij.y*rik.y + rij.z*rik.z)/(rij_norm*rik_norm);

									real P6=legendre(rdot,STEINHARDT_ORDER);

									real delta_P6=legendre_deriv(rdot,STEINHARDT_ORDER);

                  real3 delta_rijj=-delta_rik_norm/rij_norm + delta_rij_norm*rdot/rij_norm;
                  real3 delta_rikk=-delta_rij_norm/rik_norm + delta_rik_norm*rdot/rik_norm;
                  real3 delta_rijik = -delta_rijj - delta_rikk;
									Ficomp += delta_switch_ij*switch_ik*P6*delta_rij_norm + switch_ij*delta_switch_ik*P6*delta_rik_norm + switch_ij*switch_ik*delta_P6*delta_rijik;
									Fjcomp += -delta_switch_ij*switch_ik*P6*delta_rij_norm +switch_ij*switch_ik*delta_P6*delta_rijj;
									real3 Fkcomp = -switch_ij*delta_switch_ik*P6*delta_rik_norm + switch_ij*switch_ik*delta_P6*delta_rikk;
									//printf("%f %f %f\n", Fkcomp.x, Fkcomp.y, Fkcomp.z);

									atomicAdd(&F[3*k],-Fkcomp.x*M_prefactor);
									atomicAdd(&F[3*k+1],-Fkcomp.y*M_prefactor);
									atomicAdd(&F[3*k+2],-Fkcomp.z*M_prefactor);

								}
							}
						}
						//printf("%f %f %f %f %f %f\n", Fjcomp.x, Fjcomp.y, Fjcomp.z, Fjcomp2.x, Fjcomp2.y, Fjcomp2.z);
						atomicAdd(&F[3*j],-Fjcomp.x*M_prefactor-Fjcomp2.x*N_prefactor);
						atomicAdd(&F[3*j+1],-Fjcomp.y*M_prefactor-Fjcomp2.y*N_prefactor);
						atomicAdd(&F[3*j+2],-Fjcomp.z*M_prefactor-Fjcomp2.z*N_prefactor);
					}
				}
			}
			//printf("i %f %f %f\n", Ficomp.x, Ficomp.y, Ficomp.z);
			atomicAdd(&F[3*i],-Ficomp.x*M_prefactor-Ficomp2.x*N_prefactor);
			atomicAdd(&F[3*i+1],-Ficomp.y*M_prefactor-Ficomp2.y*N_prefactor);
			atomicAdd(&F[3*i+2],-Ficomp.z*M_prefactor-Ficomp2.z*N_prefactor);
			i += blockDim.x*gridDim.x;

		}



	}

extern "C" __global__ void applySteinhardtForces(int numParticles, const int* __restrict__ particles, unsigned long long* __restrict__ forceBuffers, int paddedNumAtoms, real* __restrict__ F) {

    for (int i = blockDim.x*blockIdx.x+threadIdx.x; i < numParticles; i += blockDim.x*gridDim.x) {
        int index = particles[i];
				real3 force=make_real3(F[3*i],F[3*i+1],F[3*i+2]);

        atomicAdd(&forceBuffers[index], static_cast<unsigned long long>((long long) (force.x*0x100000000)));
        atomicAdd(&forceBuffers[index+paddedNumAtoms], static_cast<unsigned long long>((long long) (force.y*0x100000000)));
        atomicAdd(&forceBuffers[index+2*paddedNumAtoms], static_cast<unsigned long long>((long long) (force.z*0x100000000)));
    }

}
